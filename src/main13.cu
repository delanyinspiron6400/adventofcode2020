#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <algorithm>
#include <limits>
#include <hip/hip_runtime_api.h>

#include "Timer.cuh"

template <int ... NUMS>
struct LoopUnrolled;

template <int OFFSET, int BUS_ID, int ... NUMS>
struct LoopUnrolled<OFFSET, BUS_ID, NUMS...>
{
	using Next = LoopUnrolled<NUMS...>;

	static constexpr __device__ bool calcResult(unsigned long long start)
	{
		return ((start + OFFSET) % BUS_ID) ? false : Next::calcResult(start);
	}
};

template <>
struct LoopUnrolled<>
{
	static constexpr __device__ bool calcResult(unsigned long long start) { return true; }
};

template <int FIRST_NUM, int ... NUMS>
__global__ void d_brute_force(unsigned long long* early_time, unsigned int* global_stop)
{
	unsigned long long start = ((threadIdx.x + blockIdx.x * blockDim.x)) * FIRST_NUM;
	for (unsigned long long iter = 0; true; ++iter, start += gridDim.x * blockDim.x * FIRST_NUM)
	{
		if (iter % 10000 == 0)
		{
			if (threadIdx.x + blockIdx.x * blockDim.x == 0)
				printf("Iteration: %llu\n", start / 1000000000ULL);
			auto remainder = atomicCAS(global_stop, 0xFFFFFFFF, 0);
			if (remainder == 1)
				return;
		}
		if(LoopUnrolled<NUMS...>::calcResult(start))
			break;
	}

	*early_time = start;
	atomicExch(global_stop, 1);
}

int main()
{
	std::cout << "Thirteenth day of Advent code!" << std::endl;
	std::fstream input("../../../data/13/input.txt", std::ios_base::in);
	if (!input.is_open())
	{
		std::cout << "Could not open file!\n";
		exit(-1);
	}

	int earliest_departure{ 0 };
	std::vector<int> buslines;
	std::vector<int> indices;
	std::vector<unsigned long long> buslines_ll;
	std::vector<std::pair<int, unsigned long long>> bus_ordering;
	input >> earliest_departure;
	std::string table_entry;
	int index{ 0 };
	while (getline(input, table_entry, ','))
	{
		if (table_entry.compare("x") != 0)
		{
			buslines.push_back(std::stoi(table_entry));
			bus_ordering.push_back(std::make_pair(index, static_cast<unsigned long long>(std::stoi(table_entry))));
			indices.push_back(index);
			buslines_ll.push_back(static_cast<unsigned long long>(std::stoi(table_entry)));
		}
		++index;
	}

	// Task 1
	int min_bus_id{std::numeric_limits<int>::max()};
	int min_add_time{ std::numeric_limits<int>::max() };
	for (auto bus : buslines)
	{
		int min_after = bus - (earliest_departure % bus);
		if (min_after < min_add_time)
		{
			min_add_time = min_after;
			min_bus_id = bus;
		}
	}

	std::cout << "Bus : " << min_bus_id << " is available " << min_add_time << " minutes after earliest depature time, so result is: " << min_add_time * min_bus_id << std::endl;

	// Task 2
	unsigned long long* d_early_time{ nullptr };
	unsigned int* d_global_stop{ nullptr };
	hipMalloc(reinterpret_cast<void**>(&d_early_time), sizeof(unsigned long long));
	hipMalloc(reinterpret_cast<void**>(&d_global_stop), sizeof(unsigned int));
	hipMemset(d_early_time, 0, sizeof(unsigned long long));
	hipMemset(d_global_stop, 0, sizeof(unsigned int));
	
	int blockSize{ 256 };
	int num_blocks_per_sm = 0;
	hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks_per_sm, d_brute_force<19, 13, 37, 19, 883, 27, 23, 32, 13, 36, 17, 50, 797, 60, 41, 79, 29>, blockSize, 0);
	int h_sm_count;
	hipDeviceGetAttribute(&h_sm_count, hipDeviceAttributeMultiprocessorCount, 0);
	int gridSize = h_sm_count* num_blocks_per_sm;
	
	unsigned long long early_time{ 0ULL };
	CPUTimer timer;
	timer.start_clock();

	d_brute_force<19, 13, 37, 19, 883, 27, 23, 32, 13, 36, 17, 50, 797, 60, 41, 79, 29> << < gridSize, blockSize >> > (d_early_time, d_global_stop);
	hipMemcpy(&early_time, d_early_time, sizeof(unsigned long long), hipMemcpyDeviceToHost);
	float timing = timer.end_clock();
	if (early_time != 0ULL)
		std::cout << "Earliest timestamp according to criterion is: " << early_time << ", found in " << timing / 1000 << " seconds" << std::endl;
	else
		std::cout << "Did not find timestamp with this datatype" << std::endl;

	return 0;
}